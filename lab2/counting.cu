#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>


void printCharArray(const char* text, int size){
    char* temp = new char[size];
    hipMemcpy(temp, text, size, hipMemcpyDeviceToHost);
    for(int i =0; i < size; ++i){
        std::cout << temp[i];
    }
    std::cout << std::endl;
    delete[] temp;
}

void printIntArray(const int* _o, int size){
    int* temp = new int[size];
    hipMemcpy(temp, _o, sizeof(int) * size, hipMemcpyDeviceToHost);
    for(int i =0; i < size; ++i){
        std::cout << temp[i] << ' ';
    }
    std::cout << std::endl;
    delete[] temp;
}

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct isAlpha {
    __host__ __device__
    int operator()(const char& x) const { return x != '\n'; }
};

void CountPosition1(const char *text, int *pos, int text_size)
{
    isAlpha op;
    const thrust::device_ptr<const char> d_text = thrust::device_pointer_cast(text);
    thrust::device_ptr<int> d_pos = thrust::device_pointer_cast(pos);

    thrust::transform(thrust::device, d_text, d_text+text_size, d_pos, op);
    thrust::inclusive_scan_by_key(thrust::device, d_pos, d_pos+text_size, d_pos, d_pos);
}

// PART II
#define BLOCKSIZE 100

__global__ void mapping(const char* text, int* pos, int text_size){
    const int index = blockIdx.x *blockDim.x + threadIdx.x;
    if (index < text_size){
        pos[index] = text[index] != '\n';
    }
}

__global__ void upSweep(int* pos, int* key, int step, int text_size, int n_op){
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= n_op) return;
    const int indLeft = index * step * 2 + (step-1);
    const int indRight = indLeft + step;
    if(indLeft >= text_size || indRight >= text_size) return;

    const int keyLeft = key[indLeft];
    const int keyRight = key[indRight];
    const int left = pos[indLeft];
    const int right = pos[indRight];


    if(keyRight == 0){
        pos[indRight] = right;
    }
    else if(keyLeft == 0){
        pos[indRight] = right + left;
        key[indRight] = 0;
    }
    else {
        pos[indRight] = left + right;    
    }
}

__global__ void downSweep(int* pos, int* key, int step, int n_op){
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= n_op) return;
    const int indLeft = index*step*2 + (step-1);
    const int indRight = indLeft + step;

    int keyLeft = key[indLeft];
    int keyRight = key[indRight];
    int left = pos[indLeft];
    int right = pos[indRight];

    if(keyLeft == 0){
        pos[indLeft] = right;
        pos[indRight] = left;
    }
    else{
        pos[indLeft] = right;
        pos[indRight] = left + right;
    }


}

void scan(int* pos, int text_size){
    int* key;

    const size_t _s = sizeof(int)*text_size;
    hipMalloc(&key, _s);
    hipMemset(key, 0, _s);
    hipMemcpy(key, pos, _s, hipMemcpyDeviceToDevice);
    
    int _step = 1;

    while(_step*2 <= text_size){
        int n_op = CeilDiv(text_size, _step*2);
        upSweep<<<CeilDiv(n_op, BLOCKSIZE), BLOCKSIZE>>>(pos, key, _step, text_size, n_op);
        _step *= 2;
    }
    if(_step == text_size) _step /= 2;

    int last;
    int* tmp;
    const int _size = (_step == text_size? _step:_step*2);
    hipMalloc(&tmp, sizeof(int)*_size);
    hipMemset(tmp, 0, sizeof(int)*_size);
    hipMemcpy(tmp, pos, sizeof(int)*text_size, hipMemcpyDeviceToDevice);
    hipMemcpy(&last, tmp+_size-1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemset(tmp+_size-1, 0, sizeof(int));


    while(_step >= 1){
        int n_op = CeilDiv(_size, _step*2);
        downSweep<<<CeilDiv(n_op, BLOCKSIZE), BLOCKSIZE>>>(tmp, key,  _step, n_op);
        _step /= 2;
    }
    
    if(_size == text_size){
        hipMemcpy(pos, tmp+1, sizeof(int)*(text_size-1), hipMemcpyDeviceToDevice);
    }
    else{
        hipMemcpy(pos, tmp+1, sizeof(int)*text_size, hipMemcpyDeviceToDevice);
    }  

    hipFree(key);
    hipFree(tmp);
}

void CountPosition2(const char *text, int *pos, int text_size)
{
    mapping<<<text_size/BLOCKSIZE + 1, BLOCKSIZE>>>(text, pos, text_size);
    scan(pos, text_size);
}
