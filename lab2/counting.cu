#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct isAlpha {
    __host__ __device__
    int operator()(const char& x) const { return x != '\n'; }
};

void CountPosition1(const char *text, int *pos, int text_size)
{
    isAlpha op;
    const thrust::device_ptr<const char> d_text = thrust::device_pointer_cast(text);
    thrust::device_ptr<int> d_pos = thrust::device_pointer_cast(pos);

    thrust::transform(thrust::device, d_text, d_text+text_size, d_pos, op);
    thrust::inclusive_scan_by_key(thrust::device, d_pos, d_pos+text_size, d_pos, d_pos);
}

// PART II
#define BLOCKSIZE 512

__global__ void mapping(const char* text, int* pos, int text_size){
    const int index = blockIdx.x *blockDim.x + threadIdx.x;
    if (index >= text_size) return;
    pos[index] = text[index] != '\n';
    
}

__global__ void upSweep(int* pos, int* key, int step, int size){
    const int tid = threadIdx.x;
    const int sid = blockIdx.x * blockDim.x;
    if(sid+tid >= size/step) return;
    const int index = (sid+tid+1)*step-1;
    if(index >= size) return;

    __shared__ int posShared[BLOCKSIZE];
    __shared__ int keyShared[BLOCKSIZE];

    posShared[tid] = pos[index];
    keyShared[tid] = key[index];
    __syncthreads();

    for(int reduce_step = 2; reduce_step <= BLOCKSIZE; reduce_step *= 2){
        if (tid % reduce_step != (reduce_step-1)) break;
        int leftid = tid-reduce_step/2;
        if(keyShared[tid] != 0){
            if(keyShared[leftid] == 0) keyShared[tid] = 0;
            posShared[tid] = posShared[tid] + posShared[leftid];
        }
        __syncthreads();
    }

    pos[index] = posShared[tid];
    key[index] = keyShared[tid];
}

__global__ void downSweep(int* pos, int* key, int step, int n_op){
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= n_op) return;
    const int indLeft = index*step*2 + (step-1);
    const int indRight = indLeft + step;

    const int keyLeft = key[indLeft];
    const int left = pos[indLeft];
    const int right = pos[indRight];

    if(keyLeft == 0){
        pos[indRight] = left;
    }
    else{
        pos[indRight] = left + right;
    }
    pos[indLeft] = right;
}


void scan(int* pos, int text_size){
    int full_size = 1;
    while(full_size < text_size){
        full_size *= 2;
    }
    
    int* tmp;
    const int _s = sizeof(int)*full_size;
    hipMalloc(&tmp, _s);
    hipMemset(tmp, 0, _s);
    hipMemcpy(tmp, pos, sizeof(int)*text_size, hipMemcpyDeviceToDevice);

    int reduce_step = BLOCKSIZE;
    while(full_size > reduce_step){
        upSweep<<<CeilDiv(text_size, reduce_step), BLOCKSIZE>>>(tmp, pos, reduce_step/BLOCKSIZE, text_size);
        reduce_step *= BLOCKSIZE;
    }
    upSweep<<<CeilDiv(text_size, reduce_step), BLOCKSIZE>>>(tmp, pos, reduce_step/BLOCKSIZE, text_size);

    int last;
    hipMemcpy(&last, tmp+full_size-1, sizeof(int), hipMemcpyDeviceToHost);

    int n_op = 0;
    reduce_step = full_size;
    while(reduce_step >= 4){
        n_op = CeilDiv(full_size, reduce_step);
        downSweep<<<CeilDiv(n_op, BLOCKSIZE), BLOCKSIZE>>>(tmp, pos, reduce_step/2, n_op);
        reduce_step /= 2;
    }
    n_op = CeilDiv(text_size, reduce_step)+1;
    downSweep<<<CeilDiv(n_op, BLOCKSIZE), BLOCKSIZE>>>(tmp, pos, reduce_step/2, n_op);

    if(full_size == text_size){
        hipMemcpy(pos, tmp+1, sizeof(int)*(text_size-1), hipMemcpyDeviceToDevice);
        hipMemset(pos+text_size-1, last, sizeof(int));
    }
    else hipMemcpy(pos, tmp+1, sizeof(int)*text_size, hipMemcpyDeviceToDevice);
    hipFree(tmp);
}

void CountPosition2(const char *text, int *pos, int text_size)
{
    mapping<<<CeilDiv(text_size, BLOCKSIZE), BLOCKSIZE>>>(text, pos, text_size);
    scan(pos, text_size);
}
